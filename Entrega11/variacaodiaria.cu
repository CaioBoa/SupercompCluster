#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>

// Função para ler os preços de ações do arquivo
void read_stock_data(const std::string& file_name, thrust::host_vector<float>& stocks) {
    std::ifstream file(file_name);
    std::string line;

    while (std::getline(file, line)) {
        float price = std::stof(line); // Converte a string para float
        stocks.push_back(price);       // Armazena no vetor
    }
}

// Functor para calcular a diferença entre dias consecutivos
struct daily_gain_functor {
    __host__ __device__
    float operator()(const float& current, const float& previous) const {
        return current - previous;  // Calcula ganho diário
    }
};

int main() {
    thrust::host_vector<float> h_stocks;
    read_stock_data("google.txt", h_stocks);

    size_t total_size = h_stocks.size();
    const size_t block_size = 100000; // Processar 100.000 elementos por vez (ajustável)

    for (size_t i = 0; i < total_size; i += block_size) {
        // Define o tamanho do bloco atual
        size_t current_block_size = std::min(block_size, total_size - i);

        // Vetores no host para o bloco atual
        thrust::host_vector<float> h_stocks_block(h_stocks.begin() + i, h_stocks.begin() + i + current_block_size);

        // Transferir o bloco para a GPU
        thrust::device_vector<float> d_stocks = h_stocks_block;

        if (d_stocks.size() > 1) {
            thrust::device_vector<float> d_daily_gain(d_stocks.size() - 1);
            thrust::transform(d_stocks.begin() + 1, d_stocks.end(), d_stocks.begin(), d_daily_gain.begin(), daily_gain_functor());

            thrust::host_vector<float> h_daily_gain = d_daily_gain;
            for (size_t j = 0; j < h_daily_gain.size(); ++j) {
                std::cout << h_daily_gain[j] << std::endl;
            }
        }
    }

    return 0;
}
