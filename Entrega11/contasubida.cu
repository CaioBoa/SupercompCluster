#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/count.h>

// Função para ler os preços de ações do arquivo
void read_stock_data(const std::string& file_name, thrust::host_vector<float>& stocks) {
    std::ifstream file(file_name);
    std::string line;

    while (std::getline(file, line)) {
        float price = std::stof(line); // Converte a string para float
        stocks.push_back(price);       // Armazena no vetor
    }
}

// Functor para calcular a diferença entre dias consecutivos
struct daily_gain_functor {
    __host__ __device__
    float operator()(const float& current, const float& previous) const {
        return current - previous;  // Calcula ganho diário
    }
};

// Functor para identificar dias de aumento no preço (diferença positiva)
struct is_positive_gain {
    __host__ __device__
    bool operator()(const float& x) const {
        return x > 0.0f;  // Retorna verdadeiro se o ganho for positivo
    }
};

int main() {
    thrust::host_vector<float> h_stocks;
    read_stock_data("google.txt", h_stocks);

    size_t total_size = h_stocks.size();
    const size_t block_size = 100000; // Tamanho do bloco a ser processado (ajustável)

    int total_positive_gains = 0;

    for (size_t i = 0; i < total_size - 1; i += block_size) {
        size_t current_block_size = std::min(block_size, total_size - i - 1);

        // Bloco atual de dados
        thrust::host_vector<float> h_stocks_block(h_stocks.begin() + i, h_stocks.begin() + i + current_block_size + 1);

        // Transferir o bloco para a GPU
        thrust::device_vector<float> d_stocks = h_stocks_block;

        if (d_stocks.size() > 1) {
            thrust::device_vector<float> d_daily_gain(d_stocks.size() - 1);
            thrust::transform(d_stocks.begin() + 1, d_stocks.end(), d_stocks.begin(), d_daily_gain.begin(), daily_gain_functor());

            // Contar quantos dias tiveram aumento no preço nesse bloco
            int num_positive_gains = thrust::count_if(d_daily_gain.begin(), d_daily_gain.end(), is_positive_gain());
            total_positive_gains += num_positive_gains;
        }
    }

    // Exibir o resultado total
    std::cout << "Número total de dias com aumento no preço: " << total_positive_gains << std::endl;

    return 0;
}
