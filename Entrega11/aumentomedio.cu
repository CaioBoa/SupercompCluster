#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/count.h>
#include <thrust/replace.h>
#include <thrust/reduce.h>

// Função para ler os preços de ações do arquivo
void read_stock_data(const std::string& file_name, thrust::host_vector<float>& stocks) {
    std::ifstream file(file_name);
    std::string line;

    while (std::getline(file, line)) {
        float price = std::stof(line); // Converte a string para float
        stocks.push_back(price);       // Armazena no vetor
    }
}

// Functor para calcular a diferença entre dias consecutivos
struct daily_gain_functor {
    __host__ __device__
    float operator()(const float& current, const float& previous) const {
        return current - previous;  // Calcula ganho diário
    }
};

// Functor para identificar dias de aumento no preço (diferença positiva)
struct is_positive_gain {
    __host__ __device__
    bool operator()(const float& x) const {
        return x > 0.0f;  // Retorna verdadeiro se o ganho for positivo
    }
};

// Functor para substituir ganhos negativos por zero
struct is_negative_gain {
    __host__ __device__
    bool operator()(const float& x) const {
        return x < 0.0f;  // Retorna verdadeiro se o ganho for negativo
    }
};

int main() {
    thrust::host_vector<float> h_stocks;
    read_stock_data("google.txt", h_stocks);

    size_t total_size = h_stocks.size();
    const size_t block_size = 100000; // Tamanho do bloco a ser processado (ajustável)

    int total_positive_gains = 0;
    float total_sum_positive_gains = 0.0f;

    for (size_t i = 0; i < total_size - 1; i += block_size) {
        size_t current_block_size = std::min(block_size, total_size - i - 1);

        // Bloco atual de dados
        thrust::host_vector<float> h_stocks_block(h_stocks.begin() + i, h_stocks.begin() + i + current_block_size + 1);

        // Transferir o bloco para a GPU
        thrust::device_vector<float> d_stocks = h_stocks_block;

        if (d_stocks.size() > 1) {
            // Calcular as diferenças diárias
            thrust::device_vector<float> d_daily_gain(d_stocks.size() - 1);
            thrust::transform(d_stocks.begin() + 1, d_stocks.end(), d_stocks.begin(), d_daily_gain.begin(), daily_gain_functor());

            // Contar quantos dias tiveram aumento no preço nesse bloco
            int num_positive_gains = thrust::count_if(d_daily_gain.begin(), d_daily_gain.end(), is_positive_gain());
            total_positive_gains += num_positive_gains;

            // Substituir valores negativos por zero
            thrust::replace_if(d_daily_gain.begin(), d_daily_gain.end(), is_negative_gain(), 0.0f);

            // Somar os valores restantes (apenas os positivos)
            float sum_positive_gains = thrust::reduce(d_daily_gain.begin(), d_daily_gain.end(), 0.0f);
            total_sum_positive_gains += sum_positive_gains;
        }
    }

    // Calcular a média dos ganhos positivos
    float average_gain = (total_positive_gains > 0) ? total_sum_positive_gains / total_positive_gains : 0.0f;

    // Exibir os resultados
    std::cout << "Número total de dias com aumento no preço: " << total_positive_gains << std::endl;
    std::cout << "Aumento médio nos dias com ganho: " << average_gain << std::endl;

    return 0;
}
